#include "hip/hip_runtime.h"
#include <nvblox/core/bounding_boxes.h>
#include <nvblox/core/unified_vector.h>
#include <nvblox/gpu_hash/cuda/gpu_hash_interface.cuh>
#include <nvblox/gpu_hash/cuda/gpu_indexing.cuh>

#include <thrust/functional.h>
#include <thrust/transform.h>

#include "nvblox_ros/conversions.hpp"

namespace nvblox {

template <typename VoxelType>
__device__ bool getVoxelIntensity(const VoxelType& voxel, float voxel_size,
                                  float* intensity);

template <>
__device__ bool getVoxelIntensity(const EsdfVoxel& voxel, float voxel_size,
                                  float* intensity) {
  *intensity = voxel_size * sqrtf(voxel.squared_distance_vox);
  if (voxel.is_inside) {
    *intensity = -*intensity;
  }
  return voxel.observed;
}

template <>
__device__ bool getVoxelIntensity(const TsdfVoxel& voxel, float voxel_size,
                                  float* intensity) {
  constexpr float kMinWeight = 0.1f;
  *intensity = voxel.distance;
  return voxel.weight > kMinWeight;
}

// Inputs: GPU hash for the E/TSDF.
//         AABB.
//         Voxel Size (just needed for ESDF).
// Outputs: vector of pcl::PointXYZIs.
//          max index (updated atomically).
template <typename VoxelType>
__global__ void copyPointcloudToPCL(
    Index3DDeviceHashMapType<VoxelBlock<VoxelType>> block_hash,
    Index3D* block_indices, size_t num_indices, int max_output_indices,
    AxisAlignedBoundingBox aabb, float block_size, PclPoint* pointcloud,
    int* max_index) {
  const float voxel_size = block_size / VoxelBlock<VoxelType>::kVoxelsPerSide;

  // Get the relevant block.
  __shared__ VoxelBlock<VoxelType>* block_ptr;
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    block_ptr = nullptr;
    auto it = block_hash.find(block_indices[blockIdx.x]);
    if (it != block_hash.end()) {
      block_ptr = it->second;
    } else {
      return;
    }
  }

  __syncthreads();

  if (block_ptr == nullptr) {
    return;
  }

  // For every voxel, check if it's in the AABB.
  Index3D voxel_index(threadIdx.x, threadIdx.y, threadIdx.z);

  // Get the voxel position:
  Vector3f voxel_position = getPositionFromBlockIndexAndVoxelIndex(
      block_size, block_indices[blockIdx.x], voxel_index);

  if (!aabb.contains(voxel_position)) {
    return;
  }

  // Check if this voxel sucks or not.
  const VoxelType& voxel =
      block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  float intensity = 0.0f;
  if (!getVoxelIntensity<VoxelType>(voxel, voxel_size, &intensity)) {
    return;
  }

  // Otherwise shove it in the output.
  int next_index = atomicAdd(max_index, 1);
  if (next_index >= max_output_indices) {
    printf("Overrunning the space. This shouldn't happen.\n");
    return;
  }
  PclPoint& point = pointcloud[next_index];
  point.x = voxel_position.x();
  point.y = voxel_position.y();
  point.z = voxel_position.z();
  point.intensity = intensity;
}

template <typename VoxelType>
void RosConverter::convertLayerInAABBToPCLCuda(
    const VoxelBlockLayer<VoxelType>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::msg::PointCloud2* pointcloud) {
  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;
  constexpr int kVoxelsPerBlock =
      kVoxelsPerSide * kVoxelsPerSide * kVoxelsPerSide;
  const float voxel_size = layer.voxel_size();

  // In case the AABB is infinite, make sure we have a finite number of
  // voxels.
  AxisAlignedBoundingBox aabb_intersect = getAABBOfAllocatedBlocks(layer);
  if (!aabb.isEmpty()) {
    aabb_intersect = aabb_intersect.intersection(aabb);
  }

  // Figure out which blocks are in the AABB.
  std::vector<Index3D> block_indices =
      getAllocatedBlocksWithinAABB(layer, aabb_intersect);
  // Copy to device memory.
  block_indices_device_ = block_indices;

  if (block_indices.empty()) {
    return;
  }
  size_t num_voxels = block_indices.size() * kVoxelsPerBlock;

  // Allocate a GPU pointcloud.
  pointcloud_device_.reserve(num_voxels);

  // Get the hash.
  GPULayerView<VoxelBlock<VoxelType>> gpu_layer_view = layer.getGpuLayerView();

  // Create an output size variable.
  if (!max_index_device_) {
    max_index_device_ = make_unified<int>(MemoryType::kDevice);
  }
  max_index_device_.setZero();

  // Call the kernel.
  int dim_block = block_indices.size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);

  copyPointcloudToPCL<VoxelType><<<dim_block, dim_threads, 0, cuda_stream_>>>(
      gpu_layer_view.getHash().impl_, block_indices_device_.data(),
      block_indices.size(), num_voxels, aabb_intersect, layer.block_size(),
      pointcloud_device_.data(), max_index_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());

  // Copy the pointcloud out.
  max_index_host_ = max_index_device_.clone(MemoryType::kHost);

  size_t output_size = sizeof(PclPoint) * *max_index_host_;
  pointcloud->data.resize(output_size);
  // Copy over all the points.
  hipMemcpy(pointcloud->data.data(), pointcloud_device_.data(), output_size,
             hipMemcpyDeviceToHost);

  // Fill the other fields in the pointcloud message.
  pointcloud->height = 1;
  pointcloud->width = *max_index_host_;
  pointcloud->point_step = sizeof(PclPoint);
  pointcloud->row_step = output_size;

  // Populate the fields.
  sensor_msgs::msg::PointField point_field;
  point_field.name = "x";
  point_field.datatype = sensor_msgs::msg::PointField::FLOAT32;
  point_field.offset = 0;
  point_field.count = 1;

  pointcloud->fields.push_back(point_field);
  point_field.name = "y";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
  point_field.name = "z";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
  point_field.name = "intensity";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
}

// Template specializations.
template void RosConverter::convertLayerInAABBToPCLCuda<TsdfVoxel>(
    const VoxelBlockLayer<TsdfVoxel>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::msg::PointCloud2* pointcloud);

template void RosConverter::convertLayerInAABBToPCLCuda<EsdfVoxel>(
    const VoxelBlockLayer<EsdfVoxel>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::msg::PointCloud2* pointcloud);

void RosConverter::meshBlockMessageFromMeshBlock(
    const MeshBlock& mesh_block, nvblox_msgs::msg::MeshBlock* mesh_block_msg) {
  CHECK_NOTNULL(mesh_block_msg);

  size_t num_vertices = mesh_block.vertices.size();

  mesh_block_msg->vertices.resize(num_vertices);
  mesh_block_msg->normals.resize(num_vertices);
  mesh_block_msg->colors.resize(mesh_block.colors.size());
  mesh_block_msg->triangles.resize(mesh_block.triangles.size());

  std::vector<Vector3f> vertices = mesh_block.getVertexVectorOnCPU();
  std::vector<Vector3f> normals = mesh_block.getNormalVectorOnCPU();
  std::vector<Color> colors = mesh_block.getColorVectorOnCPU();

  // Copy over vertices and normals.
  for (size_t i = 0; i < num_vertices; i++) {
    mesh_block_msg->vertices[i] = pointMessageFromVector(vertices[i]);
    mesh_block_msg->normals[i] = pointMessageFromVector(normals[i]);
  }

  // Copy over colors if available.
  for (size_t i = 0; i < mesh_block.colors.size(); i++) {
    mesh_block_msg->colors[i] = colorMessageFromColor(colors[i]);
  }

  // Copying over triangles is thankfully easy.
  mesh_block_msg->triangles = mesh_block.getTriangleVectorOnCPU();
}

struct DivideBy1000 : public thrust::unary_function<uint16_t, float> {
  __host__ __device__ float operator()(const uint16_t& in) {
    return static_cast<float>(in) / 1000.0f;
  }
};

// Convert image to depth frame object
bool RosConverter::depthImageFromImageMessage(
    const sensor_msgs::msg::Image::ConstSharedPtr& image_msg,
    DepthImage* depth_image) {
  CHECK_NOTNULL(depth_image);
  // If the image is a float, we can just copy it over directly.
  // If the image is int16, we need to divide by 1000 to get the correct
  // format for us.

  // First check if we actually have a valid image here.
  if (image_msg->encoding != "32FC1" && image_msg->encoding != "16UC1") {
    return false;
  }

  // Fill it in. How this is done depends on what the image encoding is.
  if (image_msg->encoding == "32FC1") {
    // Float to float, so this should be a straight-up copy. :)
    depth_image->populateFromBuffer(
        image_msg->height, image_msg->width,
        reinterpret_cast<const float*>(&image_msg->data[0]));
  } else if (image_msg->encoding == "16UC1") {
    // Then we have to just go byte-by-byte and convert this. This is a massive
    // pain and slow. We need to find a better way to do this; on GPU or
    // through openCV.
    const uint16_t* char_depth_buffer =
        reinterpret_cast<const uint16_t*>(&image_msg->data[0]);
    const int numel = image_msg->height * image_msg->width;

    bool kUseCuda = true;
    if (kUseCuda) {
      // Make sure there's enough output space.
      if (depth_image->numel() < numel) {
        *depth_image = DepthImage(image_msg->height, image_msg->width,
                                  MemoryType::kDevice);
      }

      // Now just thrust it.
      thrust::transform(char_depth_buffer, char_depth_buffer + numel,
                        depth_image->dataPtr(), DivideBy1000());
    } else {
      std::vector<float> float_depth_buffer(numel);
      for (int i = 0; i < numel; i++) {
        float_depth_buffer[i] =
            static_cast<float>(char_depth_buffer[i]) / 1000.0f;
      }
      depth_image->populateFromBuffer(image_msg->height, image_msg->width,
                                      float_depth_buffer.data(),
                                      MemoryType::kDevice);
    }
  }

  return true;
}

__global__ void populateSliceFromLayerKernel(
    Index3DDeviceHashMapType<EsdfBlock> block_hash, AxisAlignedBoundingBox aabb,
    float block_size, float* image, int rows, int cols, float z_slice_height,
    float resolution, float unobserved_value) {
  const float voxel_size = block_size / EsdfBlock::kVoxelsPerSide;
  const int pixel_col = blockIdx.x * blockDim.x + threadIdx.x;
  const int pixel_row = blockIdx.y * blockDim.y + threadIdx.y;

  if (pixel_col >= cols || pixel_row >= rows) {
    return;
  }

  // Figure out where this pixel should map to.
  Vector3f voxel_position(aabb.min().x() + resolution * pixel_col,
                          aabb.min().y() + resolution * pixel_row,
                          z_slice_height);

  Index3D block_index, voxel_index;

  getBlockAndVoxelIndexFromPositionInLayer(block_size, voxel_position,
                                           &block_index, &voxel_index);

  // Get the relevant block.
  EsdfBlock* block_ptr = nullptr;
  auto it = block_hash.find(block_index);
  if (it != block_hash.end()) {
    block_ptr = it->second;
  } else {
    image::access(pixel_row, pixel_col, cols, image) = unobserved_value;
    return;
  }

  // Get the relevant pixel.
  const EsdfVoxel* voxel =
      &block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  float distance = unobserved_value;
  if (voxel->observed) {
    distance = voxel_size * std::sqrt(voxel->squared_distance_vox);
    if (voxel->is_inside) {
      distance = -distance;
    }
  }
  image::access(pixel_row, pixel_col, cols, image) = distance;
}

void RosConverter::populateSliceFromLayer(const EsdfLayer& layer,
                                          const AxisAlignedBoundingBox& aabb,
                                          float z_slice_height,
                                          float resolution,
                                          float unobserved_value,
                                          Image<float>* image) {
  if (image->numel() <= 0) {
    return;
  }
  const float voxel_size = layer.voxel_size();

  // Create a GPU hash of the ESDF.
  GPULayerView<EsdfBlock> gpu_layer_view = layer.getGpuLayerView();

  // Pass in the GPU hash and AABB and let the kernel figure it out.
  constexpr int kThreadDim = 16;
  const int rounded_rows = static_cast<int>(
      std::ceil(image->rows() / static_cast<float>(kThreadDim)));
  const int rounded_cols = static_cast<int>(
      std::ceil(image->cols() / static_cast<float>(kThreadDim)));
  dim3 block_dim(rounded_cols, rounded_rows);
  dim3 thread_dim(kThreadDim, kThreadDim);

  populateSliceFromLayerKernel<<<block_dim, thread_dim, 0, cuda_stream_>>>(
      gpu_layer_view.getHash().impl_, aabb, layer.block_size(),
      image->dataPtr(), image->rows(), image->cols(), z_slice_height,
      resolution, unobserved_value);
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

}  // namespace nvblox
